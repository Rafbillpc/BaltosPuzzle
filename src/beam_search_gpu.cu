#include "hip/hip_runtime.h"
#include "base.hpp"
#include "beam_state.hpp"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <thrust/detail/copy.h>
#include <thrust/detail/fill.inl>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>

#define CUDA_CHECK(ans) { cuda_check((ans), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line){
  if(code != hipSuccess) {
    fprintf(stderr,"cuda: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

namespace beam_search_gpu {
  
  const u64 HASH_SIZE = 1ull<<29;
  const u64 HASH_MASK = HASH_SIZE-1;

  const u32 DEPTH_LIMIT = 1024;

  const u64 NUM_BLOCKS = 80;
  const u64 THREADS_PER_BLOCK = 512;

  const u64 NUM_THREADS = NUM_BLOCKS * THREADS_PER_BLOCK;
  const u64 TREE_SIZE_PER_THREAD = 3 << 14;

  __constant__ const u8 automaton_table[12] = {3,4,5,0,1,2,9,10,11,6,7,8};
  
  struct traverse_euler_tour_t {
    puzzle_data puzzle;
    beam_state  state;

    u32  max_score;
    u64* hash_table;
    u64* histogram;
    u32  low, high;
    
    u32 istep;
    u32 cutoff;
    f32 cutoff_keep_probability;

    u64  tour_current_total_size;
    u32* tours_current_size;
    u8*  tours_current;

    u64  tour_next_total_size;
    u32* tours_next_size;
    u8*  tours_next;

    __device__
    void run() {
      u32 idx = threadIdx.x+blockDim.x*blockIdx.x;

      u32 cutoff = this->cutoff;
      f32 cutoff_keep_probability = this->cutoff_keep_probability;
      u32 istep = this->istep;
      
      u8* tour_next = tours_next + idx * TREE_SIZE_PER_THREAD;
      u32 tour_next_size = 0;

      u64 block_size = (tour_current_total_size + NUM_THREADS - 1) / NUM_THREADS;
      u64 fr_size = idx * block_size;
      u64 to_size = min(tour_current_total_size, (idx + 1) * block_size) - 1;

      if(fr_size > to_size) {
        tours_next_size[idx] = 0;
        return;
      }

      u64 cur_size = 0;
      u32 itree = 0;
      u32 iedge = 0;

      u32 local_low = max_score, local_high = 0;

      while(cur_size + tours_current_size[itree] <= fr_size) {
        cur_size += tours_current_size[itree];
        itree += 1;
      }

      beam_state S = state;
      f32 cutoff_running = 1.0;

      u32 nstack_moves = 0;
      u8 stack_moves[DEPTH_LIMIT];
      u8 automaton_l[DEPTH_LIMIT+1];
      u8 automaton_r[DEPTH_LIMIT+1];
      automaton_l[0] = 12;
      automaton_r[0] = 12;
      
      u32 ncommit = 0;
      u8 committed[DEPTH_LIMIT];

      while(cur_size < fr_size) {
        u8* tree = tours_current + itree * TREE_SIZE_PER_THREAD;

        u8 edge = tree[iedge];
        if(edge > 0) {
          S.do_move(puzzle, edge - 1);
          if(edge <= 7) {
            automaton_l[nstack_moves+1] = automaton_table[edge - 1];
            automaton_r[nstack_moves+1] = automaton_r[nstack_moves];
          }else{
            automaton_l[nstack_moves+1] = automaton_l[nstack_moves];
            automaton_r[nstack_moves+1] = automaton_table[edge - 1];
          }
          stack_moves[nstack_moves] = edge - 1;
          nstack_moves += 1;
        }else{
          nstack_moves -= 1;
          S.undo_move(puzzle, stack_moves[nstack_moves]);
        }

        cur_size += 1;
        iedge += 1;
        while(itree < NUM_THREADS && iedge == tours_current_size[itree]) {
          itree += 1;
          iedge = 0;
        }
      }

      while(cur_size <= to_size) {

        while(cur_size <= to_size) {
          u8* tree = tours_current + itree * TREE_SIZE_PER_THREAD;

          u8 edge = tree[iedge];
          if(edge > 0) {
            S.do_move(puzzle, edge - 1);
            if(edge <= 7) {
              automaton_l[nstack_moves+1] = automaton_table[edge - 1];
              automaton_r[nstack_moves+1] = automaton_r[nstack_moves];
            }else{
              automaton_l[nstack_moves+1] = automaton_l[nstack_moves];
              automaton_r[nstack_moves+1] = automaton_table[edge - 1];
            }
            if(ncommit == nstack_moves &&
               committed[nstack_moves] == edge - 1 &&
               tour_next_size > 0) {
              stack_moves[nstack_moves] = edge - 1;
              ncommit += 1;
              nstack_moves += 1;
              tour_next_size -= 1;
            }else{
              stack_moves[nstack_moves] = edge - 1;
              nstack_moves += 1;
            }
          }else{
            if(nstack_moves == istep && fr_size <= cur_size && cur_size <= to_size) {
              break;
            }

            if(ncommit == nstack_moves) {
              tour_next[tour_next_size++] = 0;
              ncommit -= 1;
            }

            nstack_moves -= 1;
            S.undo_move(puzzle, stack_moves[nstack_moves]);
          }

          cur_size += 1;
          iedge += 1;
          while(itree < NUM_THREADS && iedge == tours_current_size[itree]) {
            itree += 1;
            iedge = 0;
          }
        }
        
        {
          if(cur_size > to_size) break;

          FOR(m, 12) if(m != automaton_l[nstack_moves] &&
                        m != automaton_r[nstack_moves]) {
            S.do_move(puzzle, m);
            u32 v = S.value(puzzle);
            bool keep = v < cutoff;
            if(v == cutoff) {
              cutoff_running += cutoff_keep_probability;
              if(cutoff_running >= 1.0) {
                cutoff_running -= 1.0;
                keep = 1;
              }
            }
            if(keep) {
              u64 h = S.get_hash(puzzle);
              u64 h_prev
                = atomicExch((unsigned long long int*)&hash_table[h&HASH_MASK],
                             h);
              if(h_prev != h) {
                while(ncommit < nstack_moves) {
                  tour_next[tour_next_size++] = 1+stack_moves[ncommit];
                  ncommit += 1;
                }
                    
                tour_next[tour_next_size++] = 1+m;
                tour_next[tour_next_size++] = 0;

                FOR(m2, 12) {
                  auto [v,h] = S.plan_move(puzzle, m2);
                  local_low = min<u32>(local_low, v);
                  local_high = max<u32>(local_high, v);
                  atomicAdd((unsigned long long int*)(histogram + v),
                            (unsigned long long int)1);
                }
              }
            }
            S.undo_move(puzzle, m);
          }
          
          if(ncommit == nstack_moves) {
            tour_next[tour_next_size++] = 0;
            ncommit -= 1;
          }

          nstack_moves -= 1;
          S.undo_move(puzzle, stack_moves[nstack_moves]);

          cur_size += 1;
          iedge += 1;
          while(itree < NUM_THREADS && iedge == tours_current_size[itree]) {
            itree += 1;
            iedge = 0;
          }
        }
      }

      while(ncommit) {
        tour_next[tour_next_size++] = 0;
        ncommit -= 1;
      }
      
      tours_next_size[idx] = tour_next_size;
     
      atomicMin(&low, local_low);
      atomicMax(&high, local_high);
      atomicAdd((unsigned long long int*)&tour_next_total_size,
                (unsigned long long int)tour_next_size);
    }
  };
    
  __global__
  __launch_bounds__(512, 2)
    void traverse_euler_tour(traverse_euler_tour_t* T) {
    T->run();
  }
  
  vector<u8> beam_search
  (puzzle_data const& puzzle,
   puzzle_state const& initial_state,
   u8 initial_direction,
   i64 width)
  {
    beam_state state; state.reset(puzzle, initial_state, initial_direction);
    i32 max_score = state.total_distance + 512;
    debug(max_score);

    traverse_euler_tour_t *traverse_data;
    CUDA_CHECK(hipMallocManaged(&traverse_data, sizeof(traverse_euler_tour_t)));

    thrust::device_vector<u64> hash_table(HASH_SIZE, 0);    
    thrust::device_vector<u64> histogram(max_score, 0);
    
    thrust::device_vector<u32> tours_current_size(NUM_THREADS, 0);    
    thrust::device_vector<u8>  tours_current(NUM_THREADS * TREE_SIZE_PER_THREAD);
    thrust::device_vector<u32> tours_next_size(NUM_THREADS, 0);    
    thrust::device_vector<u8>  tours_next(NUM_THREADS * TREE_SIZE_PER_THREAD);

    i32 cutoff = max_score;
    f32 cutoff_keep_probability = 1.0;

    u64 tour_current_total_size = 24;
    tours_current_size[0] = 24;
    FOR(m, 12) { tours_current[2*m] = 1+m; tours_current[2*m+1] = 0; }

    CUDA_CHECK(hipDeviceSynchronize());
    
    for(i32 istep = 1; istep < DEPTH_LIMIT; ++istep) {
      timer timer_s;

      traverse_data->puzzle = puzzle;
      traverse_data->state = state;

      traverse_data->max_score = max_score;
      traverse_data->hash_table
        = thrust::raw_pointer_cast(hash_table.data());
      traverse_data->histogram
        = thrust::raw_pointer_cast(histogram.data());
      traverse_data->low = max_score;
      traverse_data->high = 0;
    
      traverse_data->istep = istep;
      traverse_data->cutoff = cutoff;
      traverse_data->cutoff_keep_probability = cutoff_keep_probability;

      traverse_data->tour_current_total_size = tour_current_total_size;
      traverse_data->tours_current_size
        = thrust::raw_pointer_cast(tours_current_size.data());
      traverse_data->tours_current
        = thrust::raw_pointer_cast(tours_current.data());
      traverse_data->tour_next_total_size = 0;
      traverse_data->tours_next_size
        = thrust::raw_pointer_cast(tours_next_size.data());
      traverse_data->tours_next
        = thrust::raw_pointer_cast(tours_next.data());
    
      traverse_euler_tour<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>
        (traverse_data);
      
      CUDA_CHECK(hipPeekAtLastError());
      CUDA_CHECK(hipDeviceSynchronize());

      u32 low = traverse_data->low;
      u32 high = traverse_data->high;
      u64 tour_next_total_size = traverse_data->tour_next_total_size;

      vector<i32> L_histogram(high-low+1);
      thrust::copy(begin(histogram) + low, begin(histogram) + high+1, begin(L_histogram));
      CUDA_CHECK(hipDeviceSynchronize());
      thrust::fill(begin(histogram) + low, begin(histogram) + high+1, 0);
      CUDA_CHECK(hipDeviceSynchronize());

      { i64 total_count = 0;
        cutoff = max_score;
        cutoff_keep_probability = 1.0;
        FORU(i, low, high) {
          if(total_count + L_histogram[i-low] > width) {
            cutoff = i;
            cutoff_keep_probability
              = (float)(width-total_count) / (float)(L_histogram[i-low]);
            break;
          }
          total_count += L_histogram[i-low];
        }
      }
    
      cerr << setw(6) << (istep+2) <<
        ": scores = " << setw(3) << low << ".." << setw(3) << cutoff << ".." << setw(3) << high <<
        ", tree size = " << setw(12) << tour_next_total_size <<
        ", elapsed = " << setw(10) << timer_s.elapsed() << "s" <<
        endl;

      if(low == 0) {
        break; // TODO
      }
      
      swap(tours_current, tours_next);
      swap(tours_current_size, tours_next_size);
      tour_current_total_size = tour_next_total_size;
    }

    return {};
  }
}
