#include "hip/hip_runtime.h"
#include "beam_state.hpp"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_ptr.h>
#include <thrust/system_error.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line){
  if(code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

namespace beam_search_gpu {

  const i64 TREE_SIZE = 1<<14;
  
  const u64 HASH_SIZE = 1ull<<26;
  const u64 HASH_MASK = HASH_SIZE-1;

  using euler_tour_edge = u8;
  struct euler_tour {
    i64 max_size;
    i64 size;
    euler_tour_edge* data;
  
    FORCE_INLINE void reset() { size = 0; }
    FORCE_INLINE void push(i32 x) {
      data[size++] = x;
    }
    FORCE_INLINE u8& operator[](i32 ix) { return data[ix]; }
  };

  struct traverse_input {
    puzzle_data P;
    puzzle_state initial_state;
    u8 initial_direction;
    i32 num_tours_current;
    i32 istep;
    i32 cutoff;
    f32 cutoff_keep_probability;
  };
  
  struct traverse_output {
    i64 total_size;
    i32 next_tour_current;
    i32 num_tours_next;
    i64 low;
    i64 high;
  };

  __global__
  void traverse_euler_tour
  (traverse_input const* I,
   traverse_output* O,
   RNG* rngs,
   u64* hash_table,
   i32* tours_current_size, u8* tours_current,
   i32* tours_next_size, u8* tours_next,
   i64* histogram)
  {   
    i32* tour_next_size = 0;
    u8* tour_next = 0;

    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    RNG& rng = rngs[idx];
    
    while(1) {
      i32 ix = atomicAdd(&O->next_tour_current, 1);
      if(ix >= I->num_tours_current) break;
      
      beam_state S; S.reset(I->P, I->initial_state, I->initial_direction);
      
      i32 size = tours_current_size[ix];
      u8* tour_current = tours_current + ix * TREE_SIZE;

      i32 nstack_moves = 0;
      u8 stack_moves[MAX_SOLUTION_SIZE];
      // u32 stack_automaton[MAX_SOLUTION_SIZE];
      // stack_automaton[0] = 6*6+6;

      i32 ncommit = 0;
    
      FOR(iedge, size) {
        auto const& edge = tour_current[iedge];
        if(edge > 0) {
          stack_moves[nstack_moves] = edge - 1;
          S.do_move(I->P, stack_moves[nstack_moves]);
          // stack_automaton[nstack_moves+1]
          //   = automaton::next_state[stack_automaton[nstack_moves]][stack_moves[nstack_moves]];
          nstack_moves += 1;
        }else{
          if(nstack_moves == I->istep) {
            auto v = S.value(I->P);
            if(v < I->cutoff ||
               (v == I->cutoff && rng.randomFloat() < I->cutoff_keep_probability)) {
              if(!tour_next) {
                i32 ix2 = atomicAdd(&O->num_tours_next, 1);
                tour_next_size = tours_next_size + ix2;
                tour_next = tours_next + ix2 * TREE_SIZE;
                *tour_next_size = 0;
              }
              while(ncommit < nstack_moves) {
                tour_next[(*tour_next_size)++] = 1+stack_moves[ncommit];
                ncommit += 1;
              }

              FOR(m, 12)
                // if(automaton::allow_move[stack_automaton[istep]]&bit(m))
                {
                  auto [v,h] = S.plan_move(I->P, m);
                  auto prev
                    = atomicExch((unsigned long long int*)&hash_table[h&HASH_MASK],
                                 h);
                  if(prev != h) {
                    atomicMin((long long int*)&O->low, (long long int)v);
                    atomicMax((long long int*)&O->high, (long long int)v);
                    atomicAdd((unsigned long long int*)(histogram + v),
                              (unsigned long long int)1);
                    tour_next[(*tour_next_size)++] = 1+m;
                    tour_next[(*tour_next_size)++] = 0;
                  }
                }
            }
          }

          if(nstack_moves == 0) {
            break;
          }

          if(ncommit == nstack_moves) {
            tour_next[(*tour_next_size)++] = 0;
            ncommit -= 1;
          }

          nstack_moves -= 1;
          S.undo_move(I->P, stack_moves[nstack_moves]);
        }

        if(tour_next_size != 0 && *tour_next_size + 2 * I->istep + 128 > TREE_SIZE) {
          FORD(i,ncommit-1,0) tour_next[(*tour_next_size)++] = 0;
          tour_next[(*tour_next_size)++] = 0;

          atomicAdd((unsigned long long int*)&O->total_size,
                    (unsigned long long int)(*tour_next_size));
          tour_next_size = 0;
          tour_next = 0;
          ncommit = 0;
        }
      }
    }
      
    if(tour_next_size != 0) {
      atomicAdd((unsigned long long int*)&O->total_size,
                (unsigned long long int)(*tour_next_size));
    }
  }

  vector<u8> beam_search
  (puzzle_data const& P,
   puzzle_state const& initial_state,
   u8 initial_direction,
   i64 width)
  {
    //     if(!HS) {
    //       auto ptr = new uint64_t[HASH_SIZE];
    //       HS = ptr;
    //     }

    beam_state S; S.reset(P, initial_state, initial_direction);
    i32 max_score = S.total_distance + 1000; // TODO
    debug(max_score);

    i32 num_trees = (1<<30) / TREE_SIZE;

    i32 num_tours_current = 1;
    thrust::device_vector<u64> hash_table(HASH_SIZE);
    thrust::device_vector<i32> tours_current_size(num_trees);
    thrust::device_vector<u8>  tours_current(num_trees * TREE_SIZE);
    thrust::device_vector<i32> tours_next_size(num_trees);
    thrust::device_vector<u8>  tours_next(num_trees * TREE_SIZE);
    tours_current_size[0] = 1;
    tours_current[0] = 0;
    
    traverse_input* device_I;
    gpuErrchk(hipMallocManaged((void**)&device_I, sizeof(traverse_input)));
    traverse_output* device_O;
    gpuErrchk(hipMallocManaged((void**)&device_O, sizeof(traverse_output)));

    gpuErrchk(hipDeviceSynchronize());

    device_I->P = P;
    device_I->initial_state = initial_state;
    device_I->initial_direction = initial_direction;

    i32 cutoff = max_score;
    f32 cutoff_keep_probability = 1.0;

    i32 num_blocks  = 2048;
    i32 num_threads = 128;

    thrust::device_vector<RNG> rngs(num_blocks * num_threads);
    auto seed = time(0);
    FOR(i, num_blocks * num_threads) {
      RNG t; t.reset(seed + i);
      rngs[i] = t;
    }
    
    for(i32 istep = 0; istep < 50; ++istep) {
      timer timer_s;

      thrust::device_vector<i64> histogram(max_score+1, 0);

      device_I->num_tours_current = num_tours_current;
      device_I->istep = istep;
      device_I->cutoff = cutoff;
      device_I->cutoff_keep_probability = cutoff_keep_probability;
      device_O->total_size = 0;
      device_O->next_tour_current = 0;
      device_O->num_tours_next = 0;
      device_O->low = max_score;
      device_O->high = 0;

      gpuErrchk(hipDeviceSynchronize());
      
      traverse_euler_tour<<<num_blocks, num_threads>>>
        (device_I, device_O,
         thrust::raw_pointer_cast(rngs.data()),
         thrust::raw_pointer_cast(hash_table.data()),
         thrust::raw_pointer_cast(tours_current_size.data()),
         thrust::raw_pointer_cast(tours_current.data()),
         thrust::raw_pointer_cast(tours_next_size.data()),
         thrust::raw_pointer_cast(tours_next.data()),
         thrust::raw_pointer_cast(histogram.data())
         );
      gpuErrchk(hipPeekAtLastError());

      gpuErrchk(hipDeviceSynchronize());


      thrust::host_vector<i64> hist(histogram);

      { i64 total_count = 0;
        cutoff = max_score;
        cutoff_keep_probability = 1.0;
        FORU(i, device_O->low, device_O->high) {
          if(total_count+histogram[i] > width) {
            cutoff = i;
            cutoff_keep_probability = (float)(width-total_count) / (float)(hist[i]);
            break;
          }
          total_count += histogram[i];
        }
      }
      
      cerr << setw(6) << (istep+1) <<
        ": scores = " << setw(3) << device_O->low << ".." << setw(3) << cutoff <<
        ", tree size = " << setw(12) << device_O->total_size <<
        ", num trees = " << setw(4) << device_O->num_tours_next <<
        ", elapsed = " << setw(10) << timer_s.elapsed() << "s" <<
        endl;

      swap(tours_current, tours_next);
      swap(tours_current_size, tours_next_size);
      num_tours_current = device_O->num_tours_next;
    }

    return {};
  }

}
